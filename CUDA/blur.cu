#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include <ctime>
#include <cstdlib>
#include <iostream>

using namespace cv;
using namespace std;

//Important variables.
int rows;
int cols;
int kernel1;
int radio;
int hilos;
int bloques;
int id = 0;
Mat picture;


/*   function to compute blur   */
__global__ void blur(int *red, int *green, int *blue, int cols, int rows, int radio, int total_hi)
{
	//Index of each thread.	
	const int id_hilo = blockIdx.x * blockDim.x + threadIdx.x;
    
    //Iterate in Blockwise
    for( int i = id_hilo; i <= rows * cols; i = i+total_hi ){
        int row = (int)i / cols;
        int column = i % cols;
        int x0, x1;
        int y0, y1;

        //Check the limits - the edge pixels
        if( column - radio >= 0 ){ x0 = column - radio; }else{ x0 = 0; }
        if( column + radio < cols ){ x1 = column + radio; }else{ x1 = cols-1; }
        if( row - radio >= 0 ){ y0 = row - radio; }else{  y0 = 0; }
        if( row + radio < rows ){ y1 = row + radio; }else{ y1 = rows-1; }
        int size = (x1 - x0 + 1)*(y1 - y0 + 1);
        
        //The rgb average: rgb1 -> Red, rgb2->green, rgb3->blue
        int rgb1 = 0;
        int rgb2 = 0;
        int rgb3 = 0;

        //Calculate rgb average
        for( int i = y0; i <= y1; i++){
            for( int j = x0; j <= x1; j++ ){
                rgb1 += red[i*cols + j ];
                rgb2 += green[i*cols + j];
                rgb3 += blue[i*cols + j];
            }
        }
        
        //Assign the new values
        red[i] = (int)(rgb1 / size);
        green[i] = (int)(rgb2 / size);
        blue[i] = (int)(rgb3 / size);

    }
}

void reconstruct(int *h_red,int *h_green,int *h_blue, int cols, int rows){

    //Assign the new blured values to the picture
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++)
        {
            picture.at<Vec3b>( i, j )[0] = h_red[i*cols + j];
            picture.at<Vec3b>( i, j )[1] = h_green[i*cols + j];
            picture.at<Vec3b>( i, j )[2] = h_blue[i*cols + j];
        }
    }

}


int validations( int argc, char** argv  ){

    if( argc != 5 ){
        cout << endl;
        cout << " First parameter:  picture path with the extention ex. picture/ducati4k.jpg" << endl;
        cout << " Second parameter: kernel - between 3 and 15" << endl;
        cout << " Third parameter: threads - between 1 and 16" << endl;
        cout << " Fourth parameter: blocks - between 1 and 5" << endl;
        return -1;
    }

    picture = imread( argv[1], 1 );
    if( !picture.data ){
        cout << "No picture" << endl;
        return -1;
    }

    kernel1 = atoi( argv[2] );
    if(  kernel1 < 3 || kernel1 > 15  ){
        cout << "The kernel1 must be a number between 3 - 15" << endl;
        return -1;
    }
    hilos = atoi( argv[3] );
    bloques = atoi( argv[4] );
    
    if(  hilos % bloques != 0  ){
        cout << "The number of threads must be divisible by the number of blocks" << endl;
        return -1;
    }
    radio = int((kernel1 - 1)/2);
    return 0;
}

int main(int argc, char** argv)
{

    //Validate the input arguments
    validations(argc, argv);
    hipError_t err = hipSuccess;

    // Important Data
	int rows = picture.rows;
	int cols = picture.cols;
	int imgsize = rows * cols;
	int totalthreads = bloques * hilos;

    //Assign new host memory
	int *h_red = new int[imgsize];
	int *h_green = new int[imgsize];
	int *h_blue = new int[imgsize];
    // Define device data
    int *d_red, *d_green, *d_blue;

    //Calculate of needed space
	int colorsize = sizeof(int) * imgsize; 

    //Assign the vector rgb data into the host memory.
	for(int i = 0; i < imgsize; i++){		
        h_red[i] = (int)picture.at<Vec3b>( (int)i / cols, i % cols )[0];
        h_green[i] = (int)picture.at<Vec3b>( (int)i / cols, i % cols )[1];
        h_blue[i] = (int)picture.at<Vec3b>( (int)i / cols, i % cols )[2];
    }


	//Reserve memory in device
    err = hipMalloc<int>(&d_red, colorsize);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc<int>(&d_green, colorsize);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc<int>(&d_blue, colorsize);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


	//Copy 3 color vectors (R,G,B) from host to device
    err = hipMemcpy(d_red, h_red, colorsize, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C1 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }	
    err = hipMemcpy(d_green, h_green, colorsize, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C2 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_blue, h_blue, colorsize, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C3 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



	//Paralelize function
	blur<<<bloques, hilos>>>(d_red, d_green, d_blue, cols, rows, radio, totalthreads);
    
	err = hipDeviceSynchronize();
    if (err != hipSuccess){
        fprintf(stderr, "Failed \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_red, d_red, colorsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C1 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(h_green, d_green, colorsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C2 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(h_blue, d_blue, colorsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C3 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


	//Free device memory
    err = hipFree(d_red);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector C1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_green);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector C2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_blue);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector C3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


	//Reconstruct the new blured image
    reconstruct(h_red, h_green, h_blue,cols,rows);
	

	//Free host memory
	free(h_red);
	free(h_green);
	free(h_blue);


    //Print new blured image
    //namedWindow("picture", CV_WINDOW_NORMAL);
    //cv::imshow("picture", picture);

    //cv::waitKey();
	

	//variable to store end time

	
	//print performance information
	cout << cols << "x";
    cout << rows << "\t";
    cout << kernel1 << "\t";
	cout << hilos << "\t";
	cout << bloques << "\t";


	return 0;
}